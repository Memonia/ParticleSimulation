#include "hip/hip_runtime.h"
#include <math.h>

#include "cuda_macros.cuh"
#include "collisions.cuh"
#include "Particle.cuh"
#include "Surface.cuh"
#include "Vector.cuh"
#include "StraightLine.cuh"

void resolveCollision(Particle& p, const Surface& s)
{
	p.Velocity = p.Velocity.Reflected(
		s.LineEquation.ParallelVector.Normal()
	);
}

void resolveCollision(Particle& p1, Particle& p2)
{
	Vector v12 = p1.Velocity - p2.Velocity;
	Vector v21 = p2.Velocity - p1.Velocity;
	Vector x12 = p1.Center - p2.Center;
	Vector x21 = p2.Center - p1.Center;
	double m12 = p1.M + p2.M;
	double x12LenSqr = x12.Length * x12.Length;

	Vector nv1 = p1.Velocity - x12 * (2 * p2.M / m12) * (v12 * x12 / x12LenSqr);
	Vector nv2 = p2.Velocity - x21 * (2 * p1.M / m12) * (v21 * x21 / x12LenSqr);

	p1.Velocity = nv1;
	p2.Velocity = nv2;
}


CUDA_CALLABLE_MEMBER
double sign(double n)
{	
	return n < 0 ? -1 : 1;
}

CUDA_CALLABLE_MEMBER
double distance(const Vector& p1, const Vector& p2)
{	
	return sqrt(
		(p1.Vx - p2.Vx) * (p1.Vx - p2.Vx) + (p1.Vy - p2.Vy) * (p1.Vy - p2.Vy)
	);
}

bool isCollisionDetected(
	const Particle& p, const Surface& target, double& time_out)
{
	Vector closest{ target.LineEquation.ClosestPoint(p.Center) };
	Vector dVector{ closest - p.Center };

	if (p.Velocity * dVector <= 0)
		return false;

	double sc = target.SignedDistance(p.Center);
	double se = target.SignedDistance(p.Center + p.Velocity);
	if (sc * se > 0 && fabs(sc) > p.R && fabs(se) > p.R)
		return false;

	time_out = (sc - p.R) / (sc - se);
	return true;
}

bool isCollisionDetected(
	const Particle& p, const Particle& target, double& time_out)
{
	Vector Vab{ p.Velocity - target.Velocity };

	if (distance(p.Center, target.Center) - (p.R + target.R) > Vab.Length)
		return false;

	if (StraightLine(p.Center, Vab).Distance(target.Center) > p.R + target.R)
		return false;

	Vector dVector = p.Center - target.Center;
	if (Vab * (-dVector) <= 0)
		return false;

	double A = Vab * Vab;
	double B = dVector * Vab * 2;
	double C = dVector * dVector - (p.R + target.R) * (p.R + target.R);
	double Q = -(B + sign(B) * sqrt(B * B - 4 * A * C)) / 2;

	double t0 = Q / A;
	double t1 = C / Q;
	
	time_out = fmin(t0, t1);
	return true;
}
