#include "hip/hip_runtime.h"
#include <math.h>
#include <cstring>

#include "StraightLine.cuh"

StraightLine::StraightLine(double a, double b, double c) :
	A{ a }, B{ b }, C{ c },
	ParallelVector{ Vector(-b, a) },
	_notLine{ (a == 0) && (b == 0) }
{ }

StraightLine::StraightLine(Vector p, Vector v) :
	StraightLine
	(
		v.Vy,
		-v.Vx,
		v.Vx* p.Vy - v.Vy * p.Vx
	)
{ }

double StraightLine::Distance(Vector p) const
{
	if (!_notLine)
		return fabs(A * p.Vx + B * p.Vy + C) / sqrt(A * A + B * B);
	return NAN;
}

Vector StraightLine::ClosestPoint(Vector p) const
{
	if (_notLine)
		return Vector(0, 0);

	const double AApBB = A * A + B * B;
	double x = ((B * (B * p.Vx - A * p.Vy)) - A * C) / AApBB;
	double y = ((A * ((-B * p.Vx) + (A * p.Vy))) - B * C) / AApBB;

	return Vector(x, y);
}

Vector StraightLine::DistanceVectorFrom(Vector p) const
{
	if (!_notLine)
		return p - ClosestPoint(p);
	return Vector(0, 0);
}
