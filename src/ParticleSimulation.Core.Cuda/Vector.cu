#include "hip/hip_runtime.h"
#include <math.h>
#include <cstring>

#include "Vector.cuh"

Vector::Vector(double vx, double vy) :
	Vx{ vx },
	Vy{ vy },
	Length{ sqrt(vx * vx + vy * vy) }
{ }

Vector::Vector()
{
	std::memset(this, 0, sizeof(*this));
}

Vector Vector::Normal() const
{
	return Vector(Vy, -Vx);
}

Vector Vector::Normalised() const
{
	return Vector(Vx / Length, Vy / Length);
}

Vector Vector::Reflected(const Vector& n) const
{
	auto ned = n.Normalised();
	return *this - ned * (*this * ned) * 2;
}

Vector Vector::operator-() const
{
	return Vector(-Vx, -Vy);
}

Vector Vector::operator-(const Vector v) const
{
	return Vector(Vx - v.Vx, Vy - v.Vy);
}

Vector Vector::operator+(const Vector v) const
{
	return Vector(Vx + v.Vx, Vy + v.Vy);
}

Vector Vector::operator*(double n) const
{
	return Vector(Vx * n, Vy * n);
}

double Vector::operator*(const Vector v) const
{
	return Vx * v.Vx + Vy * v.Vy;
}
