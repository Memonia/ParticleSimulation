#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <algorithm> 

#include "hip/hip_runtime.h"
#include ""

#include "collisions.cuh"
#include "structures.cuh"

__global__
void moveParticlesKernel(
    Particle* particles, int particleCount, double time)
{
    int i = threadIdx.x;
    if (i >= particleCount)
        return;

    auto& p{ particles[i] };
    p.Center = p.Center + p.Velocity * time;
}

__global__
void nearestParticleCollisionKernel(
    Particle* particles, int particleCount, CollisionInfo* out)
{
    int j = blockIdx.x;
    int i = threadIdx.x;
    if (j >= i || i >= particleCount || j >= particleCount)
        return;

    double time;
    if (isCollisionDetected(particles[i], particles[j], time))
    {
        if (time < out[i].Time)  
        {
            out[i].Time = time;
            out[i].ParticleIndex = i;
            out[i].CollidableIndex = j;
        }
    }
}

__global__
void nearestSurfaceCollisionKernel(
    Surface* surfaces, int surfaceCount,
    Particle* particles, int particleCount, CollisionInfo* out)
{
    int j = blockIdx.x;
    int i = threadIdx.x;
    if (i >= particleCount || j >= surfaceCount)
        return;

    double time;
    if (isCollisionDetected(particles[i], surfaces[j], time))
    {
        if (time < out[i].Time)
        {
            out[i].Time = time;
            out[i].ParticleIndex = i;

            // (see CollisionInfo)
            out[i].CollidableIndex = particleCount + j;
        }
    }
}

template<typename T>
T* cudaMallocHelper(size_t allocSize)
{
    static T* ptr = 0;
    static size_t prevSize = -1;
    if (prevSize != allocSize)
    {
        prevSize = allocSize;

        hipError_t cudaStatus;
        cudaStatus = hipFree(ptr);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipFree failed!");
            return 0;
        }

        cudaStatus = hipMalloc((void**)&ptr, allocSize);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipMalloc failed!");
            return 0;
        }
    }

    return ptr;
}

hipError_t detectionKernelsHelper(const Collidables& collidables, CollisionInfo* out)
{
    size_t surfacesSize = collidables.SurfaceCount * sizeof(Surface);
    size_t particlesSize = collidables.ParticleCount * sizeof(Particle);
    size_t resultsSize = collidables.ParticleCount * sizeof(CollisionInfo);

    auto devSurfaces = cudaMallocHelper<Surface>(surfacesSize);
    auto devParticles = cudaMallocHelper<Particle>(particlesSize);
    auto devResults = cudaMallocHelper<CollisionInfo>(resultsSize);

    hipError_t cudaStatus;

    // Copy data to GPU
    cudaStatus = hipMemcpy(devResults, out, resultsSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
        goto Error;

    cudaStatus = hipMemcpy(devSurfaces, collidables.Surfaces, surfacesSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
        goto Error;

    cudaStatus = hipMemcpy(devParticles, collidables.Particles, particlesSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
        goto Error;

    // A block per surface with N threads for N particles
    nearestSurfaceCollisionKernel<<<collidables.SurfaceCount, collidables.ParticleCount>>>(
        devSurfaces, collidables.SurfaceCount, devParticles, collidables.ParticleCount, devResults
    );

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
        goto Error;

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
        goto Error;

    // A block per particle to be compared against
    nearestParticleCollisionKernel<<<collidables.ParticleCount, collidables.ParticleCount>>>(
        devParticles, collidables.ParticleCount, devResults
    );

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
        goto Error;

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
        goto Error;

    // Save results
    cudaStatus = hipMemcpy(out, devResults, resultsSize, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
        goto Error;

Return:
    return cudaStatus;

Error:
    hipFree(devResults);
    hipFree(devSurfaces);
    hipFree(devParticles);
    goto Return;
}

CollisionInfo detectNearestAndAdvanceCuda(const Collidables& collidables, double frameTime)
{
    const CollisionInfo farthest{ INFINITY, -1, -1 };
    if (collidables.ParticleCount == 0)
        return farthest;

    CollisionInfo* found;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
        goto Error;
    
    // Reuse the same memory buffer if the size didn't change
    static size_t prevCount = -1;
    static CollisionInfo* results = 0;
    static CollisionInfo* resultsEnd = 0;
    if (prevCount != collidables.ParticleCount)
    {
        prevCount = collidables.ParticleCount;

        delete[] results;
        results = new CollisionInfo[collidables.ParticleCount];
        resultsEnd = results + collidables.ParticleCount;

        // Set results to default upon creation
        std::fill(results, resultsEnd, farthest);
    }

    // Clear up from the previous call
    std::fill(results, resultsEnd, farthest);

    // Detect collisions on GPU
    cudaStatus = detectionKernelsHelper(collidables, results);
    if (cudaStatus != hipSuccess)
        goto Error;

    // Find nearest
    found = std::min_element(results, resultsEnd,
        [](auto& e1, auto& e2) { return e1.Time < e2.Time; }
    );

    if (found->Time < frameTime)
        frameTime = found->Time;

    // Move particles on GPU
    {
        auto size = collidables.ParticleCount * sizeof(Particle);

        // Particle array is still in GPU and was not overriden
        auto devParticles = cudaMallocHelper<Particle>(size);

        // A block of N threads for N particles
        moveParticlesKernel<<<1, collidables.ParticleCount>>>(
            devParticles, collidables.ParticleCount, frameTime
        );

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess)
            goto Error;

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess)
            goto Error;

        cudaStatus = hipMemcpy(collidables.Particles, devParticles, size, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
            goto Error;
    }

Return:
    return *found;

Error:
    fprintf(stderr, "Error in CUDA: %s\n", hipGetErrorString(cudaStatus));
    goto Return;
}
