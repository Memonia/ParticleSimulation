#include "hip/hip_runtime.h"
#include <math.h>

#include "cuda_macros.cuh"
#include "structures.cuh"
#include "collisions.cuh"

CUDA_CALLABLE_MEMBER
double sign(double n)
{	
	return n < 0 ? -1 : 1;
}

CUDA_CALLABLE_MEMBER
double distance(const Vector& p1, const Vector& p2)
{	
	return sqrt(
		(p1.Vx - p2.Vx) * (p1.Vx - p2.Vx) + (p1.Vy - p2.Vy) * (p1.Vy - p2.Vy)
	);
}

bool isCollisionDetected(
	const Particle& p, const Surface& target, double& time_out)
{
	Vector closest{ target.LineEquation.ClosestPoint(p.Center) };
	Vector dVector{ closest - p.Center };

	if (p.Velocity * dVector <= 0)
		return false;

	double sc = target.SignedDistance(p.Center);
	double se = target.SignedDistance(p.Center + p.Velocity);
	if (sc * se > 0 && fabs(sc) > p.R && fabs(se) > p.R)
		return false;

	time_out = (sc - p.R) / (sc - se);
	return true;
}

bool isCollisionDetected(
	const Particle& p, const Particle& target, double& time_out)
{
	Vector Vab{ p.Velocity - target.Velocity };

	if (distance(p.Center, target.Center) - (p.R + target.R) > Vab.Length)
		return false;

	if (StraightLine(p.Center, Vab).Distance(target.Center) > p.R + target.R)
		return false;

	Vector dVector = p.Center - target.Center;
	if (Vab * (-dVector) <= 0)
		return false;

	double A = Vab * Vab;
	double B = dVector * Vab * 2;
	double C = dVector * dVector - (p.R + target.R) * (p.R + target.R);
	double Q = -(B + sign(B) * sqrt(B * B - 4 * A * C)) / 2;

	double t0 = Q / A;
	double t1 = C / Q;
	
	time_out = fmin(t0, t1);
	return true;
}