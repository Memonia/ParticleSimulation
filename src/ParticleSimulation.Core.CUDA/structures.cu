#include "hip/hip_runtime.h"
#include <math.h>
#include <cstring>

#include "structures.cuh"

/* -------------------------- ------ -------------------------- */
/* -------------------------- Vector -------------------------- */
/* -------------------------- ------ -------------------------- */

Vector::Vector(double vx, double vy) :
	Vx{ vx },
	Vy{ vy },
	Length{ sqrt(vx * vx + vy * vy) }
{ }

Vector::Vector()
{
	std::memset(this, 0, sizeof(*this));
}

Vector Vector::Normal() const
{
	return Vector(Vy, -Vx);
}

Vector Vector::Normalised() const
{
	return Vector(Vx / Length, Vy / Length);
}

Vector Vector::Reflected(const Vector& n) const
{
	auto ned = n.Normalised();
	return *this - ned * (*this * ned) * 2;
}

Vector Vector::operator-() const
{
	return Vector(-Vx, -Vy);
}

Vector Vector::operator-(const Vector v) const
{
	return Vector(Vx - v.Vx, Vy - v.Vy);
}

Vector Vector::operator+(const Vector v) const
{
	return Vector(Vx + v.Vx, Vy + v.Vy);
}

Vector Vector::operator*(double n) const
{
	return Vector(Vx * n, Vy * n);
}

double Vector::operator*(const Vector v) const
{
	return Vx * v.Vx + Vy * v.Vy;
}

/* -------------------------- ------------ -------------------------- */
/* -------------------------- StraightLine -------------------------- */
/* -------------------------- ------------ -------------------------- */

StraightLine::StraightLine(double a, double b, double c) :
	A{ a }, B{ b }, C{ c },
	ParallelVector{ Vector(-b, a) },
	_notLine{ (a == 0) && (b == 0) }
{ }

StraightLine::StraightLine(Vector p, Vector v) :
	StraightLine
	(
		v.Vy,
		-v.Vx,
		v.Vx * p.Vy - v.Vy * p.Vx
	)
{ }

double StraightLine::Distance(Vector p) const
{
	if (!_notLine)
		return fabs(A * p.Vx + B * p.Vy + C) / sqrt(A * A + B * B);
	return NAN;
}

Vector StraightLine::ClosestPoint(Vector p) const
{
	if (_notLine)
		return Vector(0, 0);

	const double AApBB = A * A + B * B;
	double x = ((B * (B * p.Vx - A * p.Vy)) - A * C) / AApBB;
	double y = ((A * ((-B * p.Vx) + (A * p.Vy))) - B * C) / AApBB;

	return Vector(x, y);
}

Vector StraightLine::DistanceVectorFrom(Vector p) const
{
	if (!_notLine)
		return p - ClosestPoint(p);
	return Vector(0, 0);
}

/* -------------------------- ------- -------------------------- */
/* -------------------------- Surface -------------------------- */
/* -------------------------- ------- -------------------------- */

Surface::Surface(StraightLine lineEquation, Vector inBoundsNormal) :
	LineEquation{ lineEquation },
	InBoundsNormal{ inBoundsNormal }
{ }

double Surface::SignedDistance(const Vector p) const
{
	double d = LineEquation.Distance(p);
	Vector dVector = LineEquation.DistanceVectorFrom(p);

	if (InBoundsNormal * dVector > 0)
		return d;
	return -d;
}

/* -------------------------- -------- -------------------------- */
/* -------------------------- Particle -------------------------- */
/* -------------------------- -------- -------------------------- */

Particle::Particle(Vector center, Vector v, double r, double m) :
	Velocity{ v },
	Center{ center },
	R{ r },
	M{ m }
{ }

Particle::Particle()
{
	std::memset(this, 0, sizeof(*this));
}
